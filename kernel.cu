#include "hip/hip_runtime.h"
//kernel.cu

__global__ void
split(unsigned char *image, unsigned char *r, unsigned char *g, unsigned char *b)
{
	int p = blockIdx.x*threadnum + threadIdx.x;

	r[p] = image[p * imagechannelnum];
	g[p] = image[p * imagechannelnum + 1];
	b[p] = image[p * imagechannelnum + 2];
}
