//main.cu

        //------------------------------------import original image
	Mat originalimage;
	originalimage = imread("l0001.jpg", 1);
	namedWindow("originalimage", CV_WINDOW_AUTOSIZE);
	imshow("originalimage", originalimage);
	waitKey(0);

	//-------------------------------------save image's raw data into pointer
	unsigned char *h_imagedata = (unsigned char *)calloc(width * height * imagechannelnum, sizeof(unsigned char));
	h_imagedata = originalimage.data;

	//------------------------------------copy to device
	unsigned char *d_imagedata;
	hipMalloc(&d_imagedata, width * height * imagechannelnum);
	hipMemcpy(d_imagedata, h_imagedata, width * height * imagechannelnum, hipMemcpyHostToDevice);

	//------------------------------------declare device's outputs
	unsigned char *d_rchannel, *d_gchannel, *d_bchannel;
	hipMalloc(&d_rchannel, width * height);
	hipMalloc(&d_gchannel, width * height);
	hipMalloc(&d_bchannel, width * height);

	dim3 blocks(width * height / threadnum);
	dim3 threads(threadnum);

	//------------------------------------time and launch kernel
	float time_elapsed = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	split << <blocks, threads >> >(d_imagedata, d_rchannel, d_gchannel, d_bchannel);
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time_elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("time:%f ms\n", time_elapsed);

	//------------------------------------copy the results to host
	unsigned char *h_rchannel = (unsigned char *)calloc(width * height, sizeof(unsigned char));
	unsigned char *h_gchannel = (unsigned char *)calloc(width * height, sizeof(unsigned char));
	unsigned char *h_bchannel = (unsigned char *)calloc(width * height, sizeof(unsigned char));
	hipMemcpy(h_rchannel, d_rchannel, width * height, hipMemcpyDeviceToHost);
	hipMemcpy(h_gchannel, d_gchannel, width * height, hipMemcpyDeviceToHost);
	hipMemcpy(h_bchannel, d_bchannel, width * height, hipMemcpyDeviceToHost);

	//------------------------------------show results
	Mat red(height, width, CV_8UC1, (unsigned char*)h_rchannel);
	vector<Mat> channels;
	Mat empty;
	empty = Mat::zeros(Size(width, height), CV_8UC1);
	channels.push_back(empty);
	channels.push_back(empty);
	channels.push_back(red);
	Mat redimage;
	merge(channels, redimage);
	namedWindow("red", CV_WINDOW_AUTOSIZE);
	imshow("red", redimage);

	Mat green(height, width, CV_8UC1, (unsigned char*)h_gchannel);
	channels.clear();
	channels.push_back(empty);
	channels.push_back(green);
	channels.push_back(empty);
	Mat greenimage;
	merge(channels, greenimage);
	namedWindow("green", CV_WINDOW_AUTOSIZE);
	imshow("green", greenimage);


	Mat blue(height, width, CV_8UC1, (unsigned char*)h_bchannel);
	channels.clear();
	channels.push_back(blue);
	channels.push_back(empty);
	channels.push_back(empty);
	Mat blueimage;
	merge(channels, blueimage);
	namedWindow("blue", CV_WINDOW_AUTOSIZE);
	imshow("blue", blueimage);
	waitKey(0);
